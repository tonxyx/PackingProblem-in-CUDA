
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void binPacking (float *bins, float *items, float cap, int length)
{
  int x = 0;
  for(int i = 0; i < length; i++)
    {
      x = 0;
      if(items[i] > cap)
        {
          printf ("Element %f je veci od kapaciteta spremnika koji je %f. PREKID!\n", items[i], cap);
          break;
        }

      if(bins[0 * length + x] >= items[i])
        {
          bins[0 * length + x] -= items[i];
          bins[1 * length + x] += items[i];
        }
      else
        {
          while(bins[0 * length + x] < items[i])
            {
              x+=1;
            }
          bins[0 * length + x] -= items[i];
          bins[1 * length + x] += items[i];
          }
    }
}
